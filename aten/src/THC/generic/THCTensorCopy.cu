
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorCopy.cu"
#else

THC_API void
THCTensor_(copy)(THCState* state, THCTensor* dst, THCTensor* src) {
  if (dst == src) return;
  THC_copyTensor<THCTensor, THCTensor>(state, dst, src);
}

THC_API void
THCTensor_(copyIgnoringOverlaps)(THCState* state, THCTensor* dst, THCTensor* src) {
  // Called when we are copying into an overlapping index `dst`, but
  // we don't care which writer wins. Hacky but it works.
  // This is itself invoked by pointwiseApply2 / THCTensor_copy in
  // case that there are write overlaps.
  // FIXME: really, overlapping writes should be illegal/an error in Torch
  THC_pointwiseApply2(
    state, dst, src,
    CopyOp<typename TensorUtils<THCTensor>::DataType,
           typename TensorUtils<THCTensor>::DataType>(),
    ReadOnly, /* ignore overwrites */
    ReadOnly);
}

#define IMPLEMENT_THC_CUDA_TENSOR_COPY(TYPEC, TYPECUDA)                 \
  THC_API void                                                          \
  THCTensor_(copyCuda##TYPEC)(THCState *state,                          \
                              THCTensor *self,                          \
                              THCuda##TYPECUDA##Tensor *src) {          \
    THC_copyTensor<THCTensor, THCuda##TYPECUDA##Tensor>(state, self, src); \
  }

IMPLEMENT_THC_CUDA_TENSOR_COPY(Byte, Byte)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Char, Char)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Short, Short)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Int, Int)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Long, Long)
// THCudaTensor aka the non-existent THCudaFloatTensor
IMPLEMENT_THC_CUDA_TENSOR_COPY(Float, )
IMPLEMENT_THC_CUDA_TENSOR_COPY(Double, Double)
#ifdef CUDA_HALF_TENSOR
IMPLEMENT_THC_CUDA_TENSOR_COPY(Half, Half)
#endif

#undef IMPLEMENT_THC_CUDA_TENSOR_COPY

#endif
